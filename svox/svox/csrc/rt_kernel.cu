#include "hip/hip_runtime.h"
// The code was adapted from Yu et al. (https://github.com/sxyu/svox), published under the following license:

/*
 * Copyright 2021 PlenOctree Authors
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <cstdint>
#include <vector>
#include "common.cuh"
#include "data_spec_packed.cuh"

namespace {

// Automatically choose number of CUDA threads based on HW CUDA kernel count
int cuda_n_threads = -1;
__host__ void auto_cuda_threads() {
    if (~cuda_n_threads) return;
    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, 0);
    const int n_cores = get_sp_cores(dev_prop);
    // Optimize number of CUDA threads per block
    if (n_cores < 2048) {
        cuda_n_threads = 256;
    } if (n_cores < 8192) {
        cuda_n_threads = 512;
    } else {
        cuda_n_threads = 1;//256+128; //512;//768; //1024;
    }
}

namespace device {
// SH Coefficients from https://github.com/google/spherical-harmonics
__device__ __constant__ const float C0 = 0.28209479177387814;
__device__ __constant__ const float C1 = 0.4886025119029199;
__device__ __constant__ const float C2[] = {
    1.0925484305920792,
    -1.0925484305920792,
    0.31539156525252005,
    -1.0925484305920792,
    0.5462742152960396
};

__device__ __constant__ const float C3[] = {
    -0.5900435899266435,
    2.890611442640554,
    -0.4570457994644658,
    0.3731763325901154,
    -0.4570457994644658,
    1.445305721320277,
    -0.5900435899266435
};

__device__ __constant__ const float C4[] = {
    2.5033429417967046,
    -1.7701307697799304,
    0.9461746957575601,
    -0.6690465435572892,
    0.10578554691520431,
    -0.6690465435572892,
    0.47308734787878004,
    -1.7701307697799304,
    0.6258357354491761,
};


#define _SOFTPLUS_M1(x) (logf(1 + expf((x) - 1)))
#define _SIGMOID(x) (1 / (1 + expf(-(x))))
#define M_PIf (float)3.1415926535897932384626433832795028841971693993751058209749445923 /* pi */

template<typename scalar_t>
__host__ __device__ __inline__ static scalar_t _norm(
                scalar_t* dir) {
    return sqrtf(dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
}

template<typename scalar_t>
__host__ __device__ __inline__ static void _normalize(
                scalar_t* dir) {
    scalar_t norm = _norm(dir);
    dir[0] /= norm; dir[1] /= norm; dir[2] /= norm;
}

template<typename scalar_t>
__host__ __device__ __inline__ static scalar_t _dot3(
        const scalar_t* __restrict__ u,
        const scalar_t* __restrict__ v) {
    return u[0] * v[0] + u[1] * v[1] + u[2] * v[2];
}

__host__ __device__ __inline__ float _IDFT(
        const int i,
        const float t,
        const int T) {
    float ret;
    if (i%2 == 0) {
        ret = cosf((((float)i*M_PIf)/(float)T) * t);
    } else {
        ret = sinf(((((float)i+1)*M_PIf)/(float)T) * t);
    }
    return ret;
}

template <typename scalar_t>
__host__ __device__ __inline__ void _point2refBox(
        scalar_t* __restrict__ p,
        const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> extra_data,
        const float t,
        const int T) {
    int t0 = (int)t;
    int t1 = min(t0+1,T);
    float weight = t - t0;
    for(int i=0;i<3;++i){
        float off = extra_data[0][i] - (extra_data[0][3*t0+i] * (1.-weight) + extra_data[0][3*t1+i] * weight);
        p[i]=p[i] + off;
    }
    return;
}

// Calculate Fourier basis for given number of fourier coefficients, timestep and total number of timesteps
template <typename scalar_t>
__device__ __inline__ void precalc_fourier_basis(
        const int fourier_dim,
        float t,
        const int T,
        scalar_t* __restrict__ out) {
    for (int i = 0; i < fourier_dim; ++i) {
        out[i] = _IDFT(i,t,T);
    }
}

// Calculate basis functions depending on format, for given view directions
template <typename scalar_t>
__device__ __inline__ void maybe_precalc_basis(
    const int format,
    const int basis_dim,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        extra,
    const scalar_t* __restrict__ dir,
    scalar_t* __restrict__ out) {
    switch(format) {
        case FORMAT_ASG:
            {
                // UNTESTED ASG
                for (int i = 0; i < basis_dim; ++i) {
                    const auto& ptr = extra[i];
                    scalar_t S = _dot3(dir, &ptr[8]);
                    scalar_t dot_x = _dot3(dir, &ptr[2]);
                    scalar_t dot_y = _dot3(dir, &ptr[5]);
                    out[i] = S * expf(-ptr[0] * dot_x * dot_x
                                      -ptr[1] * dot_y * dot_y) / basis_dim;
                }
            }  // ASG
            break;
        case FORMAT_SG:
            {
                for (int i = 0; i < basis_dim; ++i) {
                    const auto& ptr = extra[i];
                    out[i] = expf(ptr[0] * (_dot3(dir, &ptr[1]) - 1.f)) / basis_dim;
                }
            }  // SG
            break;
        case FORMAT_SH:
        case FORMAT_FC: 
        case FORMAT_LFC:
            {
                out[0] = C0;
                const scalar_t x = dir[0], y = dir[1], z = dir[2];
                const scalar_t xx = x * x, yy = y * y, zz = z * z;
                const scalar_t xy = x * y, yz = y * z, xz = x * z;
                switch (basis_dim) {
                    case 25:
                        out[16] = C4[0] * xy * (xx - yy);
                        out[17] = C4[1] * yz * (3 * xx - yy);
                        out[18] = C4[2] * xy * (7 * zz - 1.f);
                        out[19] = C4[3] * yz * (7 * zz - 3.f);
                        out[20] = C4[4] * (zz * (35 * zz - 30) + 3);
                        out[21] = C4[5] * xz * (7 * zz - 3);
                        out[22] = C4[6] * (xx - yy) * (7 * zz - 1.f);
                        out[23] = C4[7] * xz * (xx - 3 * yy);
                        out[24] = C4[8] * (xx * (xx - 3 * yy) - yy * (3 * xx - yy));
                        [[fallthrough]];
                    case 16:
                        out[9] = C3[0] * y * (3 * xx - yy);
                        out[10] = C3[1] * xy * z;
                        out[11] = C3[2] * y * (4 * zz - xx - yy);
                        out[12] = C3[3] * z * (2 * zz - 3 * xx - 3 * yy);
                        out[13] = C3[4] * x * (4 * zz - xx - yy);
                        out[14] = C3[5] * z * (xx - yy);
                        out[15] = C3[6] * x * (xx - 3 * yy);
                        [[fallthrough]];
                    case 9:
                        out[4] = C2[0] * xy;
                        out[5] = C2[1] * yz;
                        out[6] = C2[2] * (2.0 * zz - xx - yy);
                        out[7] = C2[3] * xz;
                        out[8] = C2[4] * (xx - yy);
                        [[fallthrough]];
                    case 4:
                        out[1] = -C1 * y;
                        out[2] = C1 * z;
                        out[3] = -C1 * x;
                }
            }  // SH, FC, LFC
            break;

        default:
            // Do nothing
            break;
    }  // switch
}

template <typename scalar_t>
__device__ __inline__ scalar_t _get_delta_scale(
    const scalar_t* __restrict__ scaling,
    scalar_t* __restrict__ dir) {
    dir[0] *= scaling[0];
    dir[1] *= scaling[1];
    dir[2] *= scaling[2];
    scalar_t delta_scale = 1.f / _norm(dir);
    dir[0] *= delta_scale;
    dir[1] *= delta_scale;
    dir[2] *= delta_scale;
    return delta_scale;
}

template <typename scalar_t>
__device__ __inline__ void _dda_unit(
        const scalar_t* __restrict__ cen,
        const scalar_t* __restrict__ invdir,
        scalar_t* __restrict__ tmin,
        scalar_t* __restrict__ tmax) {
    // Intersect unit AABB
    scalar_t t1, t2;
    *tmin = 0.0f;
    *tmax = 1e9f;
#pragma unroll
    for (int i = 0; i < 3; ++i) {
        t1 = - cen[i] * invdir[i];
        t2 = t1 +  invdir[i];
        *tmin = max(*tmin, min(t1, t2));
        *tmax = min(*tmax, max(t1, t2));
    }
}

template <typename scalar_t>
__device__ __inline__ void trace_ray(
        PackedTreeSpec<scalar_t>& __restrict__ tree,
        SingleRaySpec<scalar_t> ray,
        RenderOptions& __restrict__ opt,
        torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> out, float timestep) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = tree.child.size(1);
    const int data_dim = tree.data.size(4);
    const int out_data_dim = out.size(0);

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = opt.background_brightness;
        }
        return;
    } else {
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = 0.f;
        }
        scalar_t pos[3];
        scalar_t basis_fn[25];
        maybe_precalc_basis<scalar_t>(opt.format, opt.basis_dim,
                tree.extra_data, ray.vdir, basis_fn);

        scalar_t fourier_basis[120]; //hardcoded size
        if (opt.format == FORMAT_FC || opt.format == FORMAT_LFC){
            if (tree.augmented_time) {
                timestep += 1.f;
                precalc_fourier_basis(max(opt.fc_dim1,opt.fc_dim2), timestep, opt.time_steps+2, fourier_basis);
            }
            else {
                precalc_fourier_basis(max(opt.fc_dim1,opt.fc_dim2), timestep, opt.time_steps, fourier_basis);
            }
        }

        scalar_t light_intensity = 1.f;
        scalar_t t = tmin;
        scalar_t cube_sz;
        const scalar_t d_rgb_pad = 1 + 2 * opt.rgb_padding;
        while (t < tmax) {
            for (int j = 0; j < 3; ++j) {
                pos[j] = ray.origin[j] + t * ray.dir[j];
            }

            int64_t node_id;
            scalar_t* tree_val = nullptr;
            
            tree_val = query_single_from_root<scalar_t>(tree.data, tree.child,
                    pos, &cube_sz, tree.weight_accum != nullptr ? &node_id : nullptr);

            scalar_t att;
            scalar_t subcube_tmin, subcube_tmax;
            _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

            const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
            const scalar_t delta_t = t_subcube + opt.step_size;
            scalar_t sigma;
            if (opt.format == FORMAT_FC){
                scalar_t tmp = 0.0;
                for (int i = 0; i < opt.fc_dim1; ++i){
                    tmp += fourier_basis[i] * tree_val[data_dim - opt.fc_dim1 + i];
                }
                sigma = tmp;
            } else if (opt.format == FORMAT_LFC){
                scalar_t tmp = 0.0;
                for (int i = 0; i < opt.fc_dim1; ++i){
                    tmp += fourier_basis[i] * tree_val[data_dim - opt.fc_dim1 + i];
                }
                sigma = expf(tmp) -1.;
            } else {
                sigma = tree_val[data_dim - 1];
            }
            if (opt.density_softplus) sigma = _SOFTPLUS_M1(sigma);
            if (sigma > opt.sigma_thresh) {
                att = expf(-delta_t * delta_scale * sigma);
                const scalar_t weight = light_intensity * (1.f - att);

                if ((opt.format != FORMAT_RGBA) && (opt.format != FORMAT_FC) && (opt.format != FORMAT_LFC)) {
                    for (int t = 0; t < out_data_dim; ++ t) {
                        int off = t * opt.basis_dim;
                        scalar_t tmp = 0.0;
                        for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                            tmp += basis_fn[i] * tree_val[off + i];
                        }
                        out[t] += weight * (_SIGMOID(tmp) * d_rgb_pad - opt.rgb_padding);
                    }
                } else if (opt.format == FORMAT_FC || opt.format == FORMAT_LFC) {
                    // fourier to sh calculation
                    scalar_t sh_tmp[3*25]; //hardcoded size
                    for (int t = 0; t < out_data_dim; ++t){
                        int off = t * opt.basis_dim * opt.fc_dim2;
                        for (int j = 0; j < opt.basis_dim; ++j){
                            scalar_t tmp = 0.0;
                            int off2 = off + j*opt.fc_dim2;
                            for (int i = 0; i < opt.fc_dim2; ++i){
                                tmp += fourier_basis[i] * tree_val[off2 +i];
                            }
                            sh_tmp[t * opt.basis_dim + j] = tmp;
                        }
                    }
                    // sh to rgb
                    for (int t = 0; t < out_data_dim; ++ t) {
                        int off = t * opt.basis_dim;
                        scalar_t tmp = 0.0;
                        for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                            tmp += basis_fn[i] * sh_tmp[off + i];
                        }
                        out[t] += weight * (_SIGMOID(tmp) * d_rgb_pad - opt.rgb_padding);
                    }
                } else {
                    for (int j = 0; j < out_data_dim; ++j) {
                        out[j] += weight * (_SIGMOID(tree_val[j]) * d_rgb_pad - opt.rgb_padding);
                    }
                }
                light_intensity *= att;

                if (tree.weight_accum != nullptr) {
                    if (tree.weight_accum_max) {
                        atomicMax(&tree.weight_accum[node_id], weight);
                    } else {
                        atomicAdd(&tree.weight_accum[node_id], weight);
                    }
                }

                if (light_intensity <= opt.stop_thresh) {
                    // Full opacity, stop
                    scalar_t scale = 1.0 / (1.0 - light_intensity);
                    for (int j = 0; j < out_data_dim; ++j) {
                        out[j] *= scale;
                    }
                    return;
                }
            }
            t += delta_t;
        }
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] += light_intensity * opt.background_brightness;
        }
    }
}

template <typename scalar_t>
__device__ __inline__ void trace_ray_backward(
    PackedTreeSpec<scalar_t>& __restrict__ tree,
    const torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t>
        grad_output,
        SingleRaySpec<scalar_t> ray,
        RenderOptions& __restrict__ opt,
    torch::PackedTensorAccessor64<scalar_t, 5, torch::RestrictPtrTraits>
        grad_data_out, float timestep) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = tree.child.size(1);
    const int data_dim = tree.data.size(4);
    const int out_data_dim = grad_output.size(0);

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        return;
    } else {
        scalar_t pos[3];
        scalar_t basis_fn[25];
        maybe_precalc_basis<scalar_t>(opt.format, opt.basis_dim, tree.extra_data,
                ray.vdir, basis_fn);

        scalar_t fourier_basis[120];
        if (opt.format == FORMAT_FC || opt.format == FORMAT_LFC){
            if (tree.augmented_time) {
                timestep += 1.f;
                precalc_fourier_basis(max(opt.fc_dim1,opt.fc_dim2), timestep, opt.time_steps+2, fourier_basis);
            }
            else {
                precalc_fourier_basis(max(opt.fc_dim1,opt.fc_dim2), timestep, opt.time_steps, fourier_basis);
            }
        }

        scalar_t accum = 0.0;
        const scalar_t d_rgb_pad = 1 + 2 * opt.rgb_padding;
        // PASS 1
        {
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) pos[j] = ray.origin[j] + t * ray.dir[j];
                scalar_t* tree_val = nullptr;
                
                tree_val = query_single_from_root<scalar_t>(tree.data, tree.child,
                        pos, &cube_sz);
                
                // Reuse offset on gradient
                const int64_t curr_leaf_offset = tree_val - tree.data.data();
                scalar_t* grad_tree_val = grad_data_out.data() + curr_leaf_offset;

                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + opt.step_size;
                scalar_t sigma;
                if (opt.format == FORMAT_FC){
                    scalar_t tmp = 0.0;
                    for (int i = 0; i < opt.fc_dim1; ++i){
                        tmp += fourier_basis[i] * tree_val[data_dim - opt.fc_dim1 + i];
                    }
                    sigma = tmp;
                } else if (opt.format == FORMAT_LFC){
                    scalar_t tmp = 0.0;
                    for (int i = 0; i < opt.fc_dim1; ++i){
                        tmp += fourier_basis[i] * tree_val[data_dim - opt.fc_dim1 + i];
                    }
                    sigma = expf(tmp) -1.;
                } else {
                    sigma = tree_val[data_dim - 1];
                }
                if (opt.density_softplus) sigma = _SOFTPLUS_M1(sigma);
                if (sigma > 0.0) {
                    att = expf(-delta_t * sigma * delta_scale);
                    const scalar_t weight = light_intensity * (1.f - att);

                    scalar_t total_color = 0.f;
                    if ((opt.format != FORMAT_RGBA) && (opt.format != FORMAT_FC) && (opt.format != FORMAT_LFC)) {
                        for (int t = 0; t < out_data_dim; ++ t) {
                            int off = t * opt.basis_dim;
                            scalar_t tmp = 0.0;
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                tmp += basis_fn[i] * tree_val[off + i];
                            }
                            const scalar_t sigmoid = _SIGMOID(tmp);
                            const scalar_t tmp2 = weight * sigmoid * (1.0 - sigmoid) *
                                                 grad_output[t] * d_rgb_pad;
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                const scalar_t toadd = basis_fn[i] * tmp2;
                                atomicAdd(&grad_tree_val[off + i],
                                        toadd);
                            }
                            total_color += (sigmoid * d_rgb_pad - opt.rgb_padding)
                                            * grad_output[t];
                        }
                    } else if (opt.format == FORMAT_FC || opt.format == FORMAT_LFC) {
                        // fourier to sh calculation
                        scalar_t sh_tmp[3*25]; //hardcoded size
                        for (int t = 0; t < out_data_dim; ++t){
                            int off = t * opt.basis_dim * opt.fc_dim2;
                            for (int j = 0; j < opt.basis_dim; ++j){
                                scalar_t tmp = 0.0;
                                int off2 = off + j*opt.fc_dim2;
                                for (int i = 0; i < opt.fc_dim2; ++i){
                                    tmp += fourier_basis[i] * tree_val[off2 +i];
                                }
                                sh_tmp[t * opt.basis_dim + j] = tmp;
                            }
                        }
                        // sh to rgb
                        for (int t = 0; t < out_data_dim; ++ t) {
                            int off = t * opt.basis_dim;
                            scalar_t tmp = 0.0;
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                tmp += basis_fn[i] * sh_tmp[off + i];
                            }
                            const scalar_t sigmoid = _SIGMOID(tmp);
                            const scalar_t tmp2 = weight * sigmoid * (1.0 - sigmoid) *
                                                 grad_output[t] * d_rgb_pad;
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                const scalar_t tmp3 = basis_fn[i] * tmp2;
                                for (int t = 0; t <= opt.fc_dim2; ++t) {
                                    scalar_t toadd = fourier_basis[t] * tmp3;
                                    atomicAdd(&grad_tree_val[opt.fc_dim2*off + opt.fc_dim2*i + t],
                                        toadd);
                                }
                            }
                            total_color += (sigmoid * d_rgb_pad - opt.rgb_padding)
                                            * grad_output[t];
                        }
                    } else {
                        for (int j = 0; j < out_data_dim; ++j) {
                            const scalar_t sigmoid = _SIGMOID(tree_val[j]);
                            const scalar_t toadd = weight * sigmoid * (
                                    1.f - sigmoid) * grad_output[j] * d_rgb_pad;
                            atomicAdd(&grad_tree_val[j], toadd);
                            total_color += (sigmoid * d_rgb_pad - opt.rgb_padding)
                                            * grad_output[j];
                        }
                    }
                    light_intensity *= att;
                    accum += weight * total_color;
                }
                t += delta_t;
            }
            scalar_t total_grad = 0.f;
            for (int j = 0; j < out_data_dim; ++j)
                total_grad += grad_output[j];
            accum += light_intensity * opt.background_brightness * total_grad;
        }
        // PASS 2
        {
            // scalar_t accum_lo = 0.0;
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) pos[j] = ray.origin[j] + t * ray.dir[j];
                scalar_t* tree_val = nullptr;

                tree_val = query_single_from_root<scalar_t>(tree.data, tree.child,
                        pos, &cube_sz);
                
                // Reuse offset on gradient
                const int64_t curr_leaf_offset = tree_val - tree.data.data();
                scalar_t* grad_tree_val = grad_data_out.data() + curr_leaf_offset;

                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + opt.step_size;
                scalar_t sigma;
                if (opt.format == FORMAT_FC){
                    scalar_t tmp = 0.0;
                    for (int i = 0; i < opt.fc_dim1; ++i){
                        tmp += fourier_basis[i] * tree_val[data_dim - opt.fc_dim1 + i];
                    }
                    sigma = tmp;
                } else if (opt.format == FORMAT_LFC) {
                    scalar_t tmp = 0.0;
                    for (int i = 0; i < opt.fc_dim1; ++i){
                        tmp += fourier_basis[i] * tree_val[data_dim - opt.fc_dim1 + i];
                    }
                    sigma = expf(tmp) -1.;
                } else {
                    sigma = tree_val[data_dim - 1];
                }
                const scalar_t raw_sigma = sigma;
                if (opt.density_softplus) sigma = _SOFTPLUS_M1(sigma);
                if (sigma > 0.0) {
                    att = expf(-delta_t * sigma * delta_scale);
                    const scalar_t weight = light_intensity * (1.f - att);

                    scalar_t total_color = 0.f;
                    if ((opt.format != FORMAT_RGBA) && (opt.format != FORMAT_FC) && (opt.format != FORMAT_LFC)) {
                        for (int t = 0; t < out_data_dim; ++ t) {
                            int off = t * opt.basis_dim;
                            scalar_t tmp = 0.0;
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                tmp += basis_fn[i] * tree_val[off + i];
                            }
                            total_color += (_SIGMOID(tmp) * d_rgb_pad - opt.rgb_padding)
                                            * grad_output[t];
                        }
                    } else if (opt.format == FORMAT_FC || opt.format == FORMAT_LFC) {
                        // fourier to sh calculation
                        scalar_t sh_tmp[3*25]; //hardcoded size
                        for (int t = 0; t < out_data_dim; ++t){
                            int off = t * opt.basis_dim * opt.fc_dim2;
                            for (int j = 0; j < opt.basis_dim; ++j){
                                scalar_t tmp = 0.0;
                                int off2 = off + j*opt.fc_dim2;
                                for (int i = 0; i < opt.fc_dim2; ++i){
                                    tmp += fourier_basis[i] * tree_val[off2 +i];
                                }
                                sh_tmp[t * opt.basis_dim + j] = tmp;
                            }
                        }
                        // sh to rgb
                        for (int t = 0; t < out_data_dim; ++ t) {
                            int off = t * opt.basis_dim;
                            scalar_t tmp = 0.0;
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                tmp += basis_fn[i] * sh_tmp[off + i];
                            }
                            total_color += (_SIGMOID(tmp) * d_rgb_pad - opt.rgb_padding)
                                            * grad_output[t];
                        }
                    } else {
                        for (int j = 0; j < out_data_dim; ++j) {
                            total_color += (_SIGMOID(tree_val[j]) * d_rgb_pad - opt.rgb_padding)
                                            * grad_output[j];
                        }
                    }
                    light_intensity *= att;
                    accum -= weight * total_color;
                    if (opt.format == FORMAT_FC) {
                        scalar_t tmp = delta_t * delta_scale * (
                                        total_color * light_intensity - accum)
                                        *  (opt.density_softplus ?
                                            _SIGMOID(raw_sigma - 1)
                                            : 1);
                        for (int t = 0; t < opt.fc_dim1; ++t) {
                            scalar_t toadd = fourier_basis[t] * tmp;
                            atomicAdd(
                                    &grad_tree_val[data_dim - opt.fc_dim1 +t],
                                    toadd
                                    );
                        }
                    } else if (opt.format == FORMAT_LFC) {
                        scalar_t tmp = delta_t * delta_scale * (
                                        total_color * light_intensity - accum)
                                        *  (opt.density_softplus ?
                                            _SIGMOID(raw_sigma - 1)
                                            : 1);
                        tmp = tmp * (raw_sigma + 1.);
                        for (int t = 0; t < opt.fc_dim1; ++t) {
                            scalar_t toadd = fourier_basis[t] * tmp;
                            atomicAdd(
                                    &grad_tree_val[data_dim - opt.fc_dim1 +t],
                                    toadd
                                    );
                        }
                    } else {
                        atomicAdd(
                            &grad_tree_val[data_dim - 1],
                            delta_t * delta_scale * (
                                total_color * light_intensity - accum)
                                *  (opt.density_softplus ?
                                    _SIGMOID(raw_sigma - 1)
                                    : 1)
                            );
                    }
                }
                t += delta_t;
            }
        }
    }
}  // trace_ray_backward

template <typename scalar_t>
__device__ __inline__ void trace_ray_se_grad_hess(
    PackedTreeSpec<scalar_t>& __restrict__ tree,
    SingleRaySpec<scalar_t> ray,
    RenderOptions& __restrict__ opt,
    torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> color_ref,
    torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> color_out,
    torch::PackedTensorAccessor64<scalar_t, 5, torch::RestrictPtrTraits>
        grad_data_out,
    torch::PackedTensorAccessor64<scalar_t, 5, torch::RestrictPtrTraits>
        hessdiag_out, float timestep) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = tree.child.size(1);
    const int data_dim = tree.data.size(4);
    const int out_data_dim = color_out.size(0);

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        for (int j = 0; j < out_data_dim; ++j) {
            color_out[j] = opt.background_brightness;
        }
        return;
    } else {
        scalar_t pos[3];
        scalar_t basis_fn[25];
        maybe_precalc_basis<scalar_t>(opt.format, opt.basis_dim, tree.extra_data,
                ray.vdir, basis_fn);

        scalar_t fourier_basis[120];
        if (opt.format == FORMAT_FC || opt.format == FORMAT_LFC){
            if (tree.augmented_time) {
                timestep += 1.f;
                precalc_fourier_basis(max(opt.fc_dim1,opt.fc_dim2), timestep, opt.time_steps+2, fourier_basis);
            }
            else {
                precalc_fourier_basis(max(opt.fc_dim1,opt.fc_dim2), timestep, opt.time_steps, fourier_basis);
            }
        }

        const scalar_t d_rgb_pad = 1 + 2 * opt.rgb_padding;

        // PASS 1 - compute residual (trace_ray_se_grad_hess)
        {
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) {
                    pos[j] = ray.origin[j] + t * ray.dir[j];
                }

                scalar_t* tree_val = nullptr;
                
                tree_val = query_single_from_root<scalar_t>(tree.data, tree.child,
                        pos, &cube_sz, nullptr);
                

                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + opt.step_size;
                scalar_t sigma;
                if (opt.format == FORMAT_FC){
                    assert(false && "FC not supported");
                } else if (opt.format == FORMAT_LFC){
                    assert(false && "LFC not supported");
                } else {
                    sigma = tree_val[data_dim - 1];
                }
                if (opt.density_softplus) sigma = _SOFTPLUS_M1(sigma);
                if (sigma > 0.0f) {
                    att = expf(-delta_t * delta_scale * sigma);
                    const scalar_t weight = light_intensity * (1.f - att);

                    if ((opt.format != FORMAT_RGBA) && (opt.format != FORMAT_FC) && (opt.format != FORMAT_LFC)) {
                        for (int t = 0; t < out_data_dim; ++ t) {
                            int off = t * opt.basis_dim;
                            scalar_t tmp = 0.0;
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                tmp += basis_fn[i] * tree_val[off + i];
                            }
                            color_out[t] += weight * (_SIGMOID(tmp) * d_rgb_pad - opt.rgb_padding);
                        }
                    } else if (opt.format == FORMAT_FC || opt.format == FORMAT_LFC) {
                        assert(false && "FC and LFC not supported");
                    } else {
                        for (int j = 0; j < out_data_dim; ++j) {
                            color_out[j] += weight * (_SIGMOID(tree_val[j]) *
                                    d_rgb_pad - opt.rgb_padding);
                        }
                    }
                    light_intensity *= att;
                }
                t += delta_t;
            }
            // Add background intensity & color -> residual
            for (int j = 0; j < out_data_dim; ++j) {
                color_out[j] += light_intensity * opt.background_brightness - color_ref[j];
            }
        }

        // PASS 2 - compute RGB gradient & suffix (trace_ray_se_grad_hess)
        scalar_t color_accum[4] = {0, 0, 0, 0};
        {
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) pos[j] = ray.origin[j] + t * ray.dir[j];
                scalar_t* tree_val = nullptr;
                
                tree_val = query_single_from_root<scalar_t>(tree.data, tree.child,
                        pos, &cube_sz);
                
                // Reuse offset on gradient
                const int64_t curr_leaf_offset = tree_val - tree.data.data();
                scalar_t* grad_tree_val = grad_data_out.data() + curr_leaf_offset;
                scalar_t* hessdiag_tree_val = hessdiag_out.data() + curr_leaf_offset;

                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + opt.step_size;
                scalar_t sigma;
                if (opt.format == FORMAT_FC){
                    assert(false && "FC not supported");
                } else if (opt.format == FORMAT_LFC){
                    assert(false && "LFC not supported");
                } else {
                    sigma = tree_val[data_dim - 1];
                }
                if (opt.density_softplus) sigma = _SOFTPLUS_M1(sigma);
                if (sigma > 0.0) {
                    att = expf(-delta_t * sigma * delta_scale);
                    const scalar_t weight = light_intensity * (1.f - att);

                    if ((opt.format != FORMAT_RGBA) && (opt.format != FORMAT_FC) && (opt.format != FORMAT_LFC)) {
                        for (int t = 0; t < out_data_dim; ++ t) {
                            int off = t * opt.basis_dim;
                            scalar_t tmp = 0.0;
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                tmp += basis_fn[i] * tree_val[off + i];
                            }
                            const scalar_t sigmoid = _SIGMOID(tmp);
                            const scalar_t grad_ci = weight * sigmoid * (1.0 - sigmoid) *
                                                  d_rgb_pad;
                            // const scalar_t d2_term =
                            //     (1.f - 2.f * sigmoid) * color_out[t];
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                const scalar_t grad_wi = basis_fn[i] * grad_ci;
                                atomicAdd(&grad_tree_val[off + i], grad_wi * color_out[t]);
                                atomicAdd(&hessdiag_tree_val[off + i],
                                        // grad_wi * basis_fn[i] * (grad_ci +
                                        //         d2_term)                   // Newton
                                        grad_wi * grad_wi                     // Gauss-Newton
                                    );
                            }
                            const scalar_t color_j = sigmoid * d_rgb_pad - opt.rgb_padding;
                            color_accum[t] += weight * color_j;
                        }
                    } else if (opt.format == FORMAT_FC) {
                        assert(false && "FC not supported");
                    } else if (opt.format == FORMAT_LFC){
                        assert(false && "LFC not supported");
                    } else {
                        for (int j = 0; j < out_data_dim; ++j) {
                            const scalar_t sigmoid = _SIGMOID(tree_val[j]);
                            const scalar_t grad_ci = weight * sigmoid * (
                                    1.f - sigmoid) * d_rgb_pad;
                            // const scalar_t d2_term = (1.f - 2.f * sigmoid) * color_out[j];
                            atomicAdd(&grad_tree_val[j], grad_ci * color_out[j]);
                            // Newton
                            // atomicAdd(&hessdiag_tree_val[j], grad_ci * (grad_ci + d2_term));
                            // Gauss-Newton
                            atomicAdd(&hessdiag_tree_val[j], grad_ci * grad_ci);
                            const scalar_t color_j = sigmoid * d_rgb_pad - opt.rgb_padding;
                            color_accum[j] += weight * color_j;
                        }
                    }
                    light_intensity *= att;
                }
                t += delta_t;
            }
            for (int j = 0; j < out_data_dim; ++j) {
                color_accum[j] += light_intensity * opt.background_brightness;
            }
        }

        // PASS 3 - finish computing sigma gradient (trace_ray_se_grad_hess)
        {
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            scalar_t color_curr[4];
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) pos[j] = ray.origin[j] + t * ray.dir[j];
                scalar_t* tree_val = nullptr;
                
                tree_val = query_single_from_root<scalar_t>(tree.data, tree.child,
                        pos, &cube_sz);

                // Reuse offset on gradient
                const int64_t curr_leaf_offset = tree_val - tree.data.data();
                scalar_t* grad_tree_val = grad_data_out.data() + curr_leaf_offset;
                scalar_t* hessdiag_tree_val = hessdiag_out.data() + curr_leaf_offset;

                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + opt.step_size;
                scalar_t sigma;
                if (opt.format == FORMAT_FC){
                    assert(false && "FC not supported");
                } else if (opt.format == FORMAT_LFC){
                    assert(false && "LFC not supported");
                } else {
                    sigma = tree_val[data_dim - 1];
                }
                const scalar_t raw_sigma = sigma;
                if (opt.density_softplus) sigma = _SOFTPLUS_M1(sigma);
                if (sigma > 0.0) {
                    att = expf(-delta_t * sigma * delta_scale);
                    const scalar_t weight = light_intensity * (1.f - att);

                    if ((opt.format != FORMAT_RGBA) && (opt.format != FORMAT_FC) && (opt.format != FORMAT_LFC)) {
                        for (int u = 0; u < out_data_dim; ++ u) {
                            int off = u * opt.basis_dim;
                            scalar_t tmp = 0.0;
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                tmp += basis_fn[i] * tree_val[off + i];
                            }
                            color_curr[u] = _SIGMOID(tmp) * d_rgb_pad - opt.rgb_padding;
                            color_accum[u] -= weight * color_curr[u];
                        }
                    } else if (opt.format == FORMAT_FC || opt.format == FORMAT_LFC) {
                        assert(false && "FC and LFC not supported");
                    } else {
                        for (int j = 0; j < out_data_dim; ++j) {
                            color_curr[j] = _SIGMOID(tree_val[j]) * d_rgb_pad - opt.rgb_padding;
                            color_accum[j] -= weight * color_curr[j];
                        }
                    }
                    light_intensity *= att;
                    for (int j = 0; j < out_data_dim; ++j) {
                        const scalar_t grad_sigma = delta_t * delta_scale * (
                                color_curr[j] * light_intensity - color_accum[j]);
                        // Newton
                        // const scalar_t grad2_sigma =
                        //     grad_sigma * (grad_sigma - delta_t * delta_scale * color_out[j]);
                        // Gauss-Newton
                        const scalar_t grad2_sigma = grad_sigma * grad_sigma;
                        if (opt.density_softplus) {
                            const scalar_t sigmoid = _SIGMOID(raw_sigma - 1);
                            const scalar_t d_sigmoid = sigmoid * (1.f - sigmoid);
                            if (opt.format == FORMAT_FC) {
                                assert(false && "FC not supported");
                            } else if (opt.format == FORMAT_LFC){
                                assert(false && "LFC not supported");
                            } else {
                                atomicAdd(&grad_tree_val[data_dim - 1], grad_sigma *
                                        color_out[j] * sigmoid);
                                atomicAdd(&hessdiag_tree_val[data_dim - 1],
                                        grad2_sigma * sigmoid * sigmoid
                                        + grad_sigma *  d_sigmoid);
                            }
                            
                        } else {
                            if (opt.format == FORMAT_FC) {
                                assert(false && "FC not supported");
                            } else if (opt.format == FORMAT_LFC){
                                assert(false && "LFC not supported");
                            } else {
                                atomicAdd(&grad_tree_val[data_dim - 1],
                                        grad_sigma * color_out[j]);
                                atomicAdd(&hessdiag_tree_val[data_dim - 1], grad2_sigma);
                            }
                            
                        }
                    }
                }
                t += delta_t;
            }
        }
        // Residual -> color
        for (int j = 0; j < out_data_dim; ++j) {
            color_out[j] += color_ref[j];
        }
    }
}

template <typename scalar_t>
__global__ void render_ray_kernel(
        PackedTreeSpec<scalar_t> tree,
        PackedRaysSpec<scalar_t> rays,
        RenderOptions opt,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        out, float timestep) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    scalar_t origin[3] = {rays.origins[tid][0], rays.origins[tid][1], rays.origins[tid][2]};

    if((opt.format == FORMAT_FC || opt.format == FORMAT_LFC) && tree.moving_cams){
        _point2refBox(origin,tree.extra_data,timestep,tree.timesteps);
    }

    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    scalar_t dir[3] = {rays.dirs[tid][0], rays.dirs[tid][1], rays.dirs[tid][2]};
    trace_ray<scalar_t>(
        tree,
        SingleRaySpec<scalar_t>{origin, dir, &rays.vdirs[tid][0]},
        opt,
        out[tid],timestep);
}


template <typename scalar_t>
__global__ void render_ray_backward_kernel(
    PackedTreeSpec<scalar_t> tree,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        grad_output,
        PackedRaysSpec<scalar_t> rays,
        RenderOptions opt,
    torch::PackedTensorAccessor64<scalar_t, 5, torch::RestrictPtrTraits>
        grad_data_out, float timestep
        ) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    scalar_t origin[3] = {rays.origins[tid][0], rays.origins[tid][1], rays.origins[tid][2]};

    if((opt.format == FORMAT_FC || opt.format == FORMAT_LFC) && tree.moving_cams){
        _point2refBox(origin,tree.extra_data,timestep,tree.timesteps);
    }

    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    scalar_t dir[3] = {rays.dirs[tid][0], rays.dirs[tid][1], rays.dirs[tid][2]};
    trace_ray_backward<scalar_t>(
        tree,
        grad_output[tid],
        SingleRaySpec<scalar_t>{origin, dir, &rays.vdirs[tid][0]},
        opt,
        grad_data_out,timestep);
}

template <typename scalar_t>
__device__ __inline__ void cam2world_ray(
    int ix, int iy,
    scalar_t* dir,
    scalar_t* origin,
    const PackedCameraSpec<scalar_t>& __restrict__ cam) {
    if (cam.K_specified) {
        scalar_t fx = cam.K[0][0];
        scalar_t fy = cam.K[1][1];
        scalar_t cx = cam.K[0][2];
        scalar_t cy = cam.K[1][2];
        scalar_t a = cam.K[0][1];
        scalar_t x = (ix / fx) - (iy * a/(fx*fy)) - (cx/fx) + (cy*a/(fx*fy));
        scalar_t y = (iy-cy)/fy;
        scalar_t z = 1;
        dir[0] = cam.c2w[0][0] * x + cam.c2w[0][1] * y + cam.c2w[0][2] * z;
        dir[1] = cam.c2w[1][0] * x + cam.c2w[1][1] * y + cam.c2w[1][2] * z;
        dir[2] = cam.c2w[2][0] * x + cam.c2w[2][1] * y + cam.c2w[2][2] * z;
        scalar_t norm = sqrtf(dir[0]*dir[0] + dir[1]*dir[1] + dir[2]*dir[2]);
        dir[0]/= norm; dir[1]/= norm; dir[2]/= norm;
        origin[0] = cam.c2w[0][3]; origin[1] = cam.c2w[1][3]; origin[2] = cam.c2w[2][3];
    } else {
        scalar_t x = (ix - 0.5 * cam.width) / cam.fx;
        scalar_t y = -(iy - 0.5 * cam.height) / cam.fy;
        scalar_t z = sqrtf(x * x + y * y + 1.0);
        x /= z; y /= z; z = -1.0f / z;
        dir[0] = cam.c2w[0][0] * x + cam.c2w[0][1] * y + cam.c2w[0][2] * z;
        dir[1] = cam.c2w[1][0] * x + cam.c2w[1][1] * y + cam.c2w[1][2] * z;
        dir[2] = cam.c2w[2][0] * x + cam.c2w[2][1] * y + cam.c2w[2][2] * z;
        origin[0] = cam.c2w[0][3]; origin[1] = cam.c2w[1][3]; origin[2] = cam.c2w[2][3];
    }
    
}


template <typename scalar_t>
__host__ __device__ __inline__ static void maybe_world2ndc(
        RenderOptions& __restrict__ opt,
        scalar_t* __restrict__ dir,
        scalar_t* __restrict__ cen, scalar_t near = 1.f) {
    if (opt.ndc_width < 0)
        return;
    scalar_t t = -(near + cen[2]) / dir[2];
    for (int i = 0; i < 3; ++i) {
        cen[i] = cen[i] + t * dir[i];
    }

    dir[0] = -((2 * opt.ndc_focal) / opt.ndc_width) * (dir[0] / dir[2] - cen[0] / cen[2]);
    dir[1] = -((2 * opt.ndc_focal) / opt.ndc_height) * (dir[1] / dir[2] - cen[1] / cen[2]);
    dir[2] = -2 * near / cen[2];

    cen[0] = -((2 * opt.ndc_focal) / opt.ndc_width) * (cen[0] / cen[2]);
    cen[1] = -((2 * opt.ndc_focal) / opt.ndc_height) * (cen[1] / cen[2]);
    cen[2] = 1 + 2 * near / cen[2];

    _normalize(dir);
}


template <typename scalar_t>
__global__ void render_image_kernel(
    PackedTreeSpec<scalar_t> tree,
    PackedCameraSpec<scalar_t> cam,
    RenderOptions opt,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        out, float timestep) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    scalar_t dir[3], origin[3];
    cam2world_ray(ix, iy, dir, origin, cam);
    maybe_world2ndc(opt, dir, origin);
    scalar_t vdir[3] = {dir[0], dir[1], dir[2]};

    if((opt.format == FORMAT_FC || opt.format == FORMAT_LFC) && tree.moving_cams){
        _point2refBox(origin,tree.extra_data,timestep,tree.timesteps);
    }

    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    trace_ray<scalar_t>(
        tree,
        SingleRaySpec<scalar_t>{origin, dir, vdir},
        opt,
        out[iy][ix], timestep);
}

template <typename scalar_t>
__global__ void render_image_backward_kernel(
    PackedTreeSpec<scalar_t> tree,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        grad_output,
    PackedCameraSpec<scalar_t> cam,
    RenderOptions opt,
    torch::PackedTensorAccessor64<scalar_t, 5, torch::RestrictPtrTraits>
        grad_data_out, float timestep) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    scalar_t dir[3], origin[3];
    cam2world_ray(ix, iy, dir, origin, cam);
    maybe_world2ndc(opt, dir, origin);
    scalar_t vdir[3] = {dir[0], dir[1], dir[2]};

    if((opt.format == FORMAT_FC || opt.format == FORMAT_LFC) && tree.moving_cams){
        _point2refBox(origin,tree.extra_data,timestep,tree.timesteps);
    }

    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    trace_ray_backward<scalar_t>(
        tree,
        grad_output[iy][ix],
        SingleRaySpec<scalar_t>{origin, dir, vdir},
        opt,
        grad_data_out, timestep);
}

template <typename scalar_t>
__global__ void se_grad_kernel(
    PackedTreeSpec<scalar_t> tree,
    PackedRaysSpec<scalar_t> rays,
    RenderOptions opt,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> color_ref,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> color_out,
    torch::PackedTensorAccessor64<scalar_t, 5, torch::RestrictPtrTraits> grad_out,
    torch::PackedTensorAccessor64<scalar_t, 5, torch::RestrictPtrTraits> hessdiag_out, float timestep) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    scalar_t origin[3] = {rays.origins[tid][0], rays.origins[tid][1], rays.origins[tid][2]};

    if((opt.format == FORMAT_FC || opt.format == FORMAT_LFC) && tree.moving_cams){
        _point2refBox(origin,tree.extra_data,timestep,tree.timesteps);
    }

    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    scalar_t dir[3] = {rays.dirs[tid][0], rays.dirs[tid][1], rays.dirs[tid][2]};

    trace_ray_se_grad_hess<scalar_t>(
        tree,
        SingleRaySpec<scalar_t>{origin, dir, &rays.vdirs[tid][0]},
        opt,
        color_ref[tid],
        color_out[tid],
        grad_out,
        hessdiag_out, timestep);
}

template <typename scalar_t>
__global__ void se_grad_persp_kernel(
    PackedTreeSpec<scalar_t> tree,
    PackedCameraSpec<scalar_t> cam,
    RenderOptions opt,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        color_ref,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        color_out,
    torch::PackedTensorAccessor64<scalar_t, 5, torch::RestrictPtrTraits> grad_out,
    torch::PackedTensorAccessor64<scalar_t, 5, torch::RestrictPtrTraits> hessdiag_out, float timestep) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    scalar_t dir[3], origin[3];
    cam2world_ray(ix, iy, dir, origin, cam);
    maybe_world2ndc(opt, dir, origin);
    scalar_t vdir[3] = {dir[0], dir[1], dir[2]};

    if((opt.format == FORMAT_FC || opt.format == FORMAT_LFC) && tree.moving_cams){
        _point2refBox(origin,tree.extra_data,timestep,tree.timesteps);
    }

    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    trace_ray_se_grad_hess<scalar_t>(
        tree,
        SingleRaySpec<scalar_t>{origin, dir, vdir},
        opt,
        color_ref[iy][ix],
        color_out[iy][ix],
        grad_out,
        hessdiag_out, timestep);
}

template <typename scalar_t>
__device__ __inline__ void grid_trace_ray(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        data,
        const scalar_t* __restrict__ origin,
        const scalar_t* __restrict__ dir,
        const scalar_t* __restrict__ vdir,
        scalar_t step_size,
        scalar_t delta_scale,
        scalar_t sigma_thresh,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        grid_weight,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        grid_hit) {
    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int reso = data.size(0);
    scalar_t* grid_weight_val = grid_weight.data();
    scalar_t* grid_hit_val = grid_hit.data();

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (dir[i] + 1e-9);
    }
    _dda_unit(origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        return;
    } else {
        scalar_t pos[3];

        scalar_t light_intensity = 1.f;
        scalar_t t = tmin;
        scalar_t cube_sz = reso;
        int32_t u, v, w, node_id;
        while (t < tmax) {
            for (int j = 0; j < 3; ++j) {
                pos[j] = origin[j] + t * dir[j];
            }

            clamp_coord<scalar_t>(pos);
            pos[0] *= reso;
            pos[1] *= reso;
            pos[2] *= reso;
            u = floor(pos[0]);
            v = floor(pos[1]);
            w = floor(pos[2]);
            pos[0] -= u;
            pos[1] -= v;
            pos[2] -= w;
            node_id = u * reso * reso + v * reso + w;

            scalar_t att;
            scalar_t subcube_tmin, subcube_tmax;
            _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

            const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
            const scalar_t delta_t = t_subcube + step_size;
            scalar_t sigma = data[u][v][w];
            sigma = expf(sigma) -1.;
            if (sigma > sigma_thresh) {
                att = expf(-delta_t * delta_scale * sigma);
                const scalar_t weight = light_intensity * (1.f - att);
                light_intensity *= att;

                atomicMax(&grid_weight_val[node_id], weight);
                atomicAdd(&grid_hit_val[node_id], (scalar_t) 1.0);
            }
            t += delta_t;
        }
    }
}

template <typename scalar_t>
__global__ void grid_weight_render_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        data,
    PackedCameraSpec<scalar_t> cam,
    RenderOptions opt,
    const scalar_t* __restrict__ offset,
    const scalar_t* __restrict__ scaling,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        grid_weight,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        grid_hit) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    scalar_t dir[3], origin[3];
    cam2world_ray(ix, iy, dir, origin, cam);
    maybe_world2ndc(opt, dir, origin);
    scalar_t vdir[3] = {dir[0], dir[1], dir[2]};

    transform_coord<scalar_t>(origin, offset, scaling);
    const scalar_t delta_scale = _get_delta_scale(scaling, dir);
    grid_trace_ray<scalar_t>(
        data,
        origin,
        dir,
        vdir,
        opt.step_size,
        delta_scale,
        opt.sigma_thresh,
        grid_weight,
        grid_hit);
}

}  // namespace device


// Compute RGB output dimension from input dimension & SH degree
__host__ int get_out_data_dim(int format, int basis_dim, int in_data_dim) {
    if ((format != FORMAT_RGBA) && (format != FORMAT_FC) && (format != FORMAT_LFC)) {
        return (in_data_dim - 1) / basis_dim;
    } else if (format == FORMAT_FC || format == FORMAT_LFC){
        return 3;
    } else {
        return in_data_dim - 1;
    }
}

}  // namespace

torch::Tensor volume_render(TreeSpec& tree, RaysSpec& rays, RenderOptions& opt, float timestep) {
    tree.check();
    rays.check();
    DEVICE_GUARD(tree.data);
    const auto Q = rays.origins.size(0);

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(opt.format, opt.basis_dim, tree.data.size(4));
    torch::Tensor result = torch::zeros({Q, out_data_dim}, rays.origins.options());
    AT_DISPATCH_FLOATING_TYPES(rays.origins.type(), __FUNCTION__, [&] {
            device::render_ray_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                    tree, rays, opt,
                    result.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(), timestep);
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor volume_render_image(TreeSpec& tree, CameraSpec& cam, RenderOptions& opt, float timestep = -1) {
    tree.check();
    cam.check();
    DEVICE_GUARD(tree.data);
    const size_t Q = size_t(cam.width) * cam.height;

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(opt.format, opt.basis_dim, tree.data.size(4));
    torch::Tensor result = torch::zeros({cam.height, cam.width, out_data_dim},
            tree.data.options());

    AT_DISPATCH_FLOATING_TYPES(tree.data.type(), __FUNCTION__, [&] {
            device::render_image_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                    tree, cam, opt,
                    result.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(), timestep);
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor volume_render_backward(
    TreeSpec& tree, RaysSpec& rays,
    RenderOptions& opt,
    torch::Tensor grad_output, float timestep) {
    tree.check();
    rays.check();
    DEVICE_GUARD(tree.data);

    const int Q = rays.origins.size(0);

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(opt.format, opt.basis_dim, tree.data.size(4));
    torch::Tensor result = torch::zeros_like(tree.data);
    AT_DISPATCH_FLOATING_TYPES(rays.origins.type(), __FUNCTION__, [&] {
            device::render_ray_backward_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                tree,
                grad_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                rays,
                opt,
                result.packed_accessor64<scalar_t, 5, torch::RestrictPtrTraits>(), timestep);
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor volume_render_image_backward(TreeSpec& tree, CameraSpec& cam,
                                           RenderOptions& opt,
                                           torch::Tensor grad_output, float timestep) {
    tree.check();
    cam.check();
    DEVICE_GUARD(tree.data);

    const size_t Q = size_t(cam.width) * cam.height;

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(opt.format, opt.basis_dim, tree.data.size(4));
    torch::Tensor result = torch::zeros_like(tree.data);

    AT_DISPATCH_FLOATING_TYPES(tree.data.type(), __FUNCTION__, [&] {
            device::render_image_backward_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                tree,
                grad_output.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                cam,
                opt,
                result.packed_accessor64<scalar_t, 5, torch::RestrictPtrTraits>(), timestep);
    });
    CUDA_CHECK_ERRORS;
    return result;
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> se_grad(
        TreeSpec& tree, RaysSpec& rays, torch::Tensor color, RenderOptions& opt, float timestep) {
    tree.check();
    rays.check();
    DEVICE_GUARD(tree.data);
    CHECK_INPUT(color);

    const auto Q = rays.origins.size(0);

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(opt.format, opt.basis_dim, tree.data.size(4));
    if (out_data_dim > 4) {
        throw std::runtime_error("Tree's output dim cannot be > 4 for se_grad");
    }
    torch::Tensor result = torch::zeros({Q, out_data_dim}, rays.origins.options());
    torch::Tensor grad = torch::zeros_like(tree.data);
    torch::Tensor hessdiag = torch::zeros_like(tree.data);
    AT_DISPATCH_FLOATING_TYPES(rays.origins.type(), __FUNCTION__, [&] {
            device::se_grad_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                    tree, rays, opt,
                    color.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    result.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    grad.packed_accessor64<scalar_t, 5, torch::RestrictPtrTraits>(),
                    hessdiag.packed_accessor64<scalar_t, 5, torch::RestrictPtrTraits>(), timestep);
    });
    CUDA_CHECK_ERRORS;
    return std::template tuple<torch::Tensor, torch::Tensor, torch::Tensor>(result, grad, hessdiag);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> se_grad_persp(
                            TreeSpec& tree,
                            CameraSpec& cam,
                            RenderOptions& opt,
                            torch::Tensor color, float timestep) {
    tree.check();
    cam.check();
    DEVICE_GUARD(tree.data);
    CHECK_INPUT(color);
    const size_t Q = size_t(cam.width) * cam.height;

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(opt.format, opt.basis_dim, tree.data.size(4));
    if (out_data_dim > 4) {
        throw std::runtime_error("Tree's output dim cannot be > 4 for se_grad");
    }
    torch::Tensor result = torch::zeros({cam.height, cam.width, out_data_dim},
            tree.data.options());
    torch::Tensor grad = torch::zeros_like(tree.data);
    torch::Tensor hessdiag = torch::zeros_like(tree.data);

    AT_DISPATCH_FLOATING_TYPES(tree.data.type(), __FUNCTION__, [&] {
            device::se_grad_persp_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                    tree, cam, opt,
                    color.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                    result.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                    grad.packed_accessor64<scalar_t, 5, torch::RestrictPtrTraits>(),
                    hessdiag.packed_accessor64<scalar_t, 5, torch::RestrictPtrTraits>(), timestep);
    });
    CUDA_CHECK_ERRORS;
    return std::template tuple<torch::Tensor, torch::Tensor, torch::Tensor>(result, grad, hessdiag);
}
std::vector<torch::Tensor> grid_weight_render(
    torch::Tensor data, CameraSpec& cam, RenderOptions& opt,
    torch::Tensor offset, torch::Tensor scaling) {
    cam.check();
    DEVICE_GUARD(data);
    const size_t Q = size_t(cam.width) * cam.height;

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    torch::Tensor grid_weight = torch::zeros_like(data);
    torch::Tensor grid_hit = torch::zeros_like(data);

    AT_DISPATCH_FLOATING_TYPES(data.type(), __FUNCTION__, [&] {
            device::grid_weight_render_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                data.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                cam,
                opt,
                offset.data<scalar_t>(),
                scaling.data<scalar_t>(),
                grid_weight.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                grid_hit.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return {grid_weight, grid_hit};
}
