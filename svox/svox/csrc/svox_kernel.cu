#include "hip/hip_runtime.h"
// The code was adapted from Yu et al. (https://github.com/sxyu/svox), published under the following license:

/*
 * Copyright 2021 PlenOctree Authors
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <cstdint>
#include "common.cuh"
#include "data_spec_packed.cuh"

#define CUDA_N_THREADS 1024

namespace {
void check_indices(torch::Tensor& indices) {
    CHECK_INPUT(indices);
    TORCH_CHECK(indices.dim() == 2);
    TORCH_CHECK(indices.is_floating_point());
}

namespace device {

template <typename scalar_t>
__device__ __inline__ scalar_t* get_tree_leaf_ptr(
       torch::PackedTensorAccessor64<scalar_t, 5, torch::RestrictPtrTraits>
        data,
       PackedTreeSpec<scalar_t>& __restrict__ tree,
       const scalar_t* __restrict__ xyz_ind,
       int64_t* node_id=nullptr) {
    scalar_t xyz[3] = {xyz_ind[0], xyz_ind[1], xyz_ind[2]};
    transform_coord<scalar_t>(xyz, tree.offset, tree.scaling);
    scalar_t _cube_sz;
    return query_single_from_root<scalar_t>(data, tree.child,
            xyz, &_cube_sz, node_id);
}

template <typename scalar_t>
__global__ void query_single_kernel(
        PackedTreeSpec<scalar_t> tree,
        const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> indices,
        torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> values_out,
        torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> node_ids_out) {
    CUDA_GET_THREAD_ID(tid, indices.size(0));
    scalar_t* data_ptr = get_tree_leaf_ptr(tree.data, tree, &indices[tid][0], &node_ids_out[tid]);
    for (int i = 0; i < tree.data.size(4); ++i)
        values_out[tid][i] = data_ptr[i];
}

template <typename scalar_t>
__global__ void query_single_kernel_backward(
       PackedTreeSpec<scalar_t> tree,
       const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> indices,
       const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> grad_output,
       torch::PackedTensorAccessor64<scalar_t, 5, torch::RestrictPtrTraits> grad_data_out) {
    CUDA_GET_THREAD_ID(tid, indices.size(0));
    scalar_t* data_ptr = get_tree_leaf_ptr(grad_data_out, tree, &indices[tid][0]);
    for (int i = 0; i < grad_output.size(1); ++i)
        atomicAdd(&data_ptr[i], grad_output[tid][i]);
}

template <typename scalar_t>
__global__ void assign_single_kernel(
       PackedTreeSpec<scalar_t> tree,
       const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> indices,
       const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> values) {
    CUDA_GET_THREAD_ID(tid, indices.size(0));
    scalar_t* data_ptr = get_tree_leaf_ptr(tree.data, tree, &indices[tid][0]);
    for (int i = 0; i < values.size(1); ++i)
        data_ptr[i] = values[tid][i];
}


template <typename scalar_t>
__global__ void calc_corner_kernel(
       PackedTreeSpec<scalar_t> tree,
       const torch::PackedTensorAccessor32<int64_t, 2, torch::RestrictPtrTraits> indexer,
       torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output) {
    CUDA_GET_THREAD_ID(tid, indexer.size(0));
    const int N = tree.data.size(1);
    const auto* leaf = &indexer[tid][0];
    scalar_t* result = &output[tid][0];

    int32_t curr[4] = {(int32_t) leaf[0], (int32_t) leaf[1],
                       (int32_t) leaf[2], (int32_t) leaf[3]};
    while (true) {
        for (int i = 0; i < 3; ++i) {
            result[i] += curr[i + 1];
            result[i] /= N;
        }
        if (curr[0] == 0) break;
        curr[0] = tree.parent_depth[curr[0]][0];
        for (int i = 3; i > 0; --i) {
            curr[i] = curr[0] % N;
            curr[0] /= N;
        }
    }
}

}  // namespace device
}  // namespace

QueryResult query_vertical(TreeSpec& tree, torch::Tensor indices) {
    tree.check();
    check_indices(indices);
    DEVICE_GUARD(indices);

    const auto Q = indices.size(0), K = tree.data.size(4);

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, CUDA_N_THREADS);
    torch::Tensor values = torch::empty({Q, K}, indices.options());
    auto node_ids_options = at::TensorOptions()
                       .dtype(at::kLong)
                       .layout(tree.child.layout())
                       .device(tree.child.device());
    torch::Tensor node_ids = torch::empty({Q}, node_ids_options);
    AT_DISPATCH_FLOATING_TYPES(indices.type(), __FUNCTION__, [&] {
        device::query_single_kernel<scalar_t><<<blocks, CUDA_N_THREADS>>>(
                tree,
                indices.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                values.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                node_ids.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return QueryResult(values, node_ids);
}

void assign_vertical(TreeSpec& tree, torch::Tensor indices, torch::Tensor values) {
    tree.check();
    check_indices(indices);
    check_indices(values);
    DEVICE_GUARD(indices);
    const int blocks = CUDA_N_BLOCKS_NEEDED(indices.size(0), CUDA_N_THREADS);
    AT_DISPATCH_FLOATING_TYPES(indices.type(), __FUNCTION__, [&] {
        device::assign_single_kernel<scalar_t><<<blocks, CUDA_N_THREADS>>>(
                tree,
                indices.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                values.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
}

torch::Tensor query_vertical_backward(
        TreeSpec& tree,
        torch::Tensor indices,
        torch::Tensor grad_output) {
    tree.check();
    DEVICE_GUARD(indices);
    const auto Q = indices.size(0), N = tree.child.size(1),
               K = grad_output.size(1), M = tree.child.size(0);
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, CUDA_N_THREADS);

    torch::Tensor grad_data = torch::zeros({M, N, N, N, K}, grad_output.options());

    AT_DISPATCH_FLOATING_TYPES(indices.type(), __FUNCTION__, [&] {
        device::query_single_kernel_backward<scalar_t><<<blocks, CUDA_N_THREADS>>>(
                tree,
                indices.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                grad_output.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                grad_data.packed_accessor64<scalar_t, 5, torch::RestrictPtrTraits>());
    });

    CUDA_CHECK_ERRORS;
    return grad_data;
}

torch::Tensor calc_corners(
        TreeSpec& tree,
        torch::Tensor indexer) {
    tree.check();
    DEVICE_GUARD(indexer);
    const auto Q = indexer.size(0);
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, CUDA_N_THREADS);

    torch::Tensor output = torch::zeros({Q, 3}, tree.data.options());

    AT_DISPATCH_FLOATING_TYPES(tree.data.type(), __FUNCTION__, [&] {
        device::calc_corner_kernel<scalar_t><<<blocks, CUDA_N_THREADS>>>(
                tree,
                indexer.packed_accessor32<int64_t, 2, torch::RestrictPtrTraits>(),
                output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    });

    CUDA_CHECK_ERRORS;
    return output;
}
